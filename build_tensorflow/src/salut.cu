#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void kernel(int *g_data) {
    const unsigned int idx = threadIdx.x;
    int data = g_data[idx];
    g_data[idx] = ((((data <<  0) >> 24) + 10) << 24)
                  | ((((data <<  8) >> 24) + 10) << 16)
                  | ((((data << 16) >> 24) + 10) <<  8)
                  | ((((data << 24) >> 24) + 10) <<  0);

}

int main() {
    const size_t len = 512;
    const unsigned int num_threads = len / 4;
    using byte = unsigned char;
    const auto mem_size = sizeof(byte) * len;
    constexpr byte init_val = 32;
    std::vector<byte> host_vec(len, init_val);
    byte *gpu_data;

    checkCudaErrors(hipMalloc((void **) &gpu_data, mem_size));
    checkCudaErrors(hipMemcpy(gpu_data, &host_vec[0], mem_size, hipMemcpyHostToDevice));

    kernel<<<1, num_threads>>>((int *) gpu_data);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipMemcpy(&host_vec[0], gpu_data, mem_size, hipMemcpyDeviceToHost));
    for (auto i = 0; i < len; ++i) {
        int val = static_cast<int>(host_vec[i]);
        if (val != init_val + 10) {
            std::cerr << "FAILED: idx " << i
                      << " val = " << val << " != " << init_val + 10 << std::endl;
        }
    }
}